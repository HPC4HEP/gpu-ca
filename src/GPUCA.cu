#include "hip/hip_runtime.h"
/*
 * GPUCA.cu
 *
 *  Created on: Jun 17, 2015
 *      Author: fpantale
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */
#include <vector>
#include "Cell.h"
#include "CUDAQueue.h"
#include "SimpleHit.h"
#include "PacketHeader.h"
#include <iostream>
#include "eclipse_parser.h"
#include <assert.h>     /* assert */


// Maximum relative difference (par1_A - par1_B)/par1_A for each parameters
constexpr float c_maxDoubletRelDifference[]{0.1, 0.1};
constexpr int c_doubletParametersNum = sizeof(c_maxDoubletRelDifference)/sizeof(c_maxDoubletRelDifference[0]);
constexpr int c_maxCellsNumPerLayer  = 256;
constexpr int c_maxNeighborsNumPerCell = 32;

template <int maxNumLayersInPacket>
__inline__
__device__
int getNumHitsInLayer(const PacketHeader<maxNumLayersInPacket>* __restrict__ packetHeader, const int layer )
{
	int numHitsInLayer = 0;
	if(layer < packetHeader->numLayers)
	{
		numHitsInLayer = (layer == packetHeader->numLayers -1) ?
				packetHeader->size - packetHeader->firstHitIdOnLayer[layer]:
				packetHeader->firstHitIdOnLayer[layer+1] - packetHeader->firstHitIdOnLayer[layer];
	}

	return numHitsInLayer;


}


__inline__
__device__
bool isADoublet(const SimpleHit* __restrict__ hits, const int idOrigin, const int idTarget)
{
	float relEtaDiff = 2*fabs((hits[idOrigin].eta - hits[idTarget].eta)/(hits[idOrigin].eta+hits[idTarget].eta));
	if(relEtaDiff > c_maxDoubletRelDifference[0]) return false;
	float relPhiDiff = 2*fabs((hits[idOrigin].phi - hits[idTarget].phi)/(hits[idOrigin].phi+hits[idTarget].phi));
	if(relPhiDiff > c_maxDoubletRelDifference[1]) return false;

	return true;
}


// this will become a global kernel in the offline CA
template< int maxNumLayersInPacket,int maxCellsNum, int warpSize >
__device__ void makeCells (const PacketHeader<maxNumLayersInPacket>* __restrict__ packetHeader, const SimpleHit* __restrict__ hits,
		CUDAQueue<maxCellsNum,Cell<c_maxNeighborsNumPerCell, c_doubletParametersNum> >& outputCells,int hitId )
{
	auto threadInWarpIdx = threadIdx.x%32;
	auto layerId = hits[hitId].layerId;
	auto firstHitIdOnNextLayer = packetHeader->firstHitIdOnLayer[layerId+1];
	auto numHitsOnNextLayer = getNumHitsInLayer(packetHeader, layerId+1 );
	auto nSteps = (numHitsOnNextLayer+warpSize-1)/warpSize;
	for (auto i = 0; i < nSteps; ++i)
	{
		auto targetHitId = i*warpSize + threadInWarpIdx;
		if(targetHitId < numHitsOnNextLayer)
		{
			if(isADoublet(hits, hitId, targetHitId))
			{
				auto cellId = outputCells.push(Cell<c_maxNeighborsNumPerCell, c_doubletParametersNum>(hitId, targetHitId, layerId, outputCells.m_data));
				if(cellId == -1)
					break;

			}

		}

	}

}


template <int maxNumLayersInPacket, int maxCellsNum, int maxNeighborsNumPerCell, int doubletParametersNum>
__global__ void singleBlockCA (const PacketHeader<maxNumLayersInPacket>* __restrict__ packetHeader, const SimpleHit* __restrict__ packetPayload )
{
	auto warpIdx = (blockDim.x*blockIdx.x + threadIdx.x)/32;
	auto warpNum = blockDim.x/32;
	auto threadInWarpIdx = threadIdx.x%32;
	__shared__ CUDAQueue<maxCellsNum, Cell<maxNeighborsNumPerCell, doubletParametersNum> > foundCells;

	//We will now create cells with the inner hit on each layer except the last one, which does not have a layer next to it.
	auto numberOfOriginHitsInInnerLayers = packetHeader->firstHitIdOnLayer[packetHeader->numLayers-1];

	auto nSteps = (numberOfOriginHitsInInnerLayers+warpNum-1)/warpNum;


	for (auto i = 0; i < nSteps; ++i)
	{
		auto hitIdx = warpIdx + warpNum*i;
		if(hitIdx < numberOfOriginHitsInInnerLayers)
		{
			makeCells (packetHeader, packetPayload, foundCells, hitIdx);
		}


	}
	__syncthreads();





}




int main()
{
	constexpr auto numLayers = 5;
	constexpr auto numHitsPerLayer = 100;

	srand (time(NULL));
	std::pair<float, float> range_eta(0.1, 0.3);
	std::pair<float, float> range_phi(0.5, 0.6);

	std::vector<SimpleHit> hitsVector(numLayers*numHitsPerLayer);



	for (auto i = 0; i< numLayers; ++i)
	{
		for(auto j =0; j<numHitsPerLayer; ++j)
		{
			hitsVector[i*numHitsPerLayer + j].eta = range_eta.first + (range_eta.second - range_eta.first)*(static_cast <float> (rand()) / static_cast <float> (RAND_MAX));
			hitsVector[i*numHitsPerLayer + j].phi = range_phi.first + (range_phi.second - range_phi.first)*(static_cast <float> (rand()) / static_cast <float> (RAND_MAX));
			hitsVector[i*numHitsPerLayer + j].layerId = i;
			std::cout << i*numHitsPerLayer + j << " "<<  hitsVector[i*numHitsPerLayer + j].eta << " " << hitsVector[i*numHitsPerLayer + j].phi << " " << hitsVector[i*numHitsPerLayer + j].layerId << std::endl;

		}
	}


	int* host_Packet;
	int* device_Packet;
	auto packetSize = sizeof(PacketHeader<c_maxNumberOfLayersInPacket>) + hitsVector.size()*sizeof(SimpleHit);
	hipHostMalloc((void**)&host_Packet, packetSize);
	hipMalloc((void**)&device_Packet, packetSize);
	PacketHeader<c_maxNumberOfLayersInPacket>* host_packetHeader = (PacketHeader<c_maxNumberOfLayersInPacket>*)(host_Packet);
	SimpleHit* host_packetPayload = (SimpleHit*)((char*)host_Packet + sizeof(PacketHeader<c_maxNumberOfLayersInPacket>));


	//initialization of the Packet to send to the GPU
	host_packetHeader->size = hitsVector.size();
	host_packetHeader->numLayers = numLayers;
	for(auto i = 0; i<numLayers; ++i)
		host_packetHeader->firstHitIdOnLayer[i] = i*numHitsPerLayer;
	memcpy(host_packetPayload, hitsVector.data(), hitsVector.size()*sizeof(SimpleHit));

	// end of the initialization



	for (auto i = 0; i< numLayers; ++i)
	{
		for(auto j =0; j<numHitsPerLayer; ++j)
		{
			assert(hitsVector[i*numHitsPerLayer + j].eta == host_packetPayload[i*numHitsPerLayer + j].eta);
			assert(hitsVector[i*numHitsPerLayer + j].phi == host_packetPayload[i*numHitsPerLayer + j].phi);
			assert(hitsVector[i*numHitsPerLayer + j].layerId == host_packetPayload[i*numHitsPerLayer + j].layerId);

		}
	}
	hipMemcpyAsync(device_Packet, host_Packet, packetSize, hipMemcpyHostToDevice, 0);

	singleBlockCA<c_maxNumberOfLayersInPacket,  c_maxCellsNumPerLayer*c_maxNumberOfLayersInPacket,c_maxNeighborsNumPerCell , c_doubletParametersNum><<<1,2048>>>(
			device_Packet, (SimpleHit*)((char*)device_Packet+sizeof(host_packetHeader)));





	hipHostFree(host_Packet);
	hipFree(device_Packet);

}
